#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cmath>

#include <sys/time.h>

#include <thrust/device_vector.h>

#include "spin_stepper.hpp"
#include "fourier_analyzer.hpp"
#include "spin_initializer.hpp"

typedef double value_type;
typedef thrust::device_vector< value_type > device_type;

typedef std::vector< value_type > host_type;

double time_diff_in_ms( timeval &t1 , timeval &t2 )
{ return (t2.tv_sec - t1.tv_sec)*1000.0 + (t2.tv_usec - t1.tv_usec)/1000.0 + 0.5; }

void spin_relax( const int N , const double J , const double q , 
                 const int steps, const double dt , 
                 const double beta0 , const double nu )
{
    const double delta_sqr = 1 - 2*J*J;
    std::clog << "System size: " << N << ", sigma: " << delta_sqr << std::endl;


    /* define initial conditions */
    host_type s_x_host( N+2 , 0.0 );
    host_type s_y_host( N+2 , 0.0 );
    host_type s_z_host( N+2 , 0.0 );

    host_type h_x_host( N , 1.0 );
    host_type h_y_host( N , 1.0 );
    host_type h_z_host( N , 1.0 );

    /* some test intial conditions */
    /*
      s_x_host[ N/2+1 ] = sqrt(0.5);
      s_y_host[ N/2+1 ] = sqrt(0.5);
      s_x_host[ N/2+2 ] = 1.0;
    */

    spin_initializer< value_type > init( q , nu );

    init.init_gaussian_random( delta_sqr , h_x_host , h_y_host , h_z_host );
    //init.init_normalized_random( h_x_host , h_y_host , h_z_host );
    init.init_normalized_random( s_x_host , s_y_host , s_z_host );

    // fix edges to zero:
    s_x_host[0] = 0.0; s_x_host[N+1] = 0.0;
    s_y_host[0] = 0.0; s_y_host[N+1] = 0.0;
    s_z_host[0] = 0.0; s_z_host[N+1] = 0.0;

    init.relax( h_x_host , h_y_host , h_z_host , 
                s_x_host , s_y_host , s_z_host ,
                beta0 );
    
    //std::clog << "initialization finished" << std::endl;
    
    // initialize device vectors
    //std::clog << "copy to device..." << std::endl;
    
    // vectors s_*_host have length N+2 to account for boundary conditions
    device_type s_x( s_x_host.begin() , s_x_host.end() );
    device_type s_y( s_y_host.begin() , s_y_host.end() );
    device_type s_z( s_z_host.begin() , s_z_host.end() );
    
    device_type h_x( h_x_host.begin() , h_x_host.end() );
    device_type h_y( h_y_host.begin() , h_y_host.end() );
    device_type h_z( h_z_host.begin() , h_z_host.end() );
    
    device_type energies( N );
    
    spin_stepper< device_type , value_type > stepper( N , J , h_x , h_y , h_z );
    fourier_analyzer< device_type , value_type > fourier( N , q );
    
    stepper.energies( s_x , s_y , s_z , energies );
    
    char filename[255];
    sprintf( filename , "../data_study_N%d/fourier_J%.5f_q%.5f.dat" , N , J , q/N );
    std::ofstream res_file( filename );
    if( !res_file.good() )
    {
        std::cout << "unable to open file: " << filename << std::endl;
        exit(-1);
    }

    //std::clog << "Starting time evolution..." << std::endl;

    timeval elapsed_time_start , elapsed_time_end;
    gettimeofday(&elapsed_time_start , NULL);
	
    int n;
    
    for( n=0 ; n<steps ; ++n )
    {
        
        
        if( (n%10) == 0 )
        {
            res_file << n*dt << '\t';
            stepper.energies( s_x , s_y , s_z , energies );
            res_file << thrust::reduce( energies.begin() , energies.end() ) << '\t';
            const double fq = fourier.analyze( energies )/N;
            res_file << fq << std::endl;
            if( fq < 1.0 ) 
                break;
        }
        
        stepper.do_step( s_x , s_y , s_z , dt );
    }

    gettimeofday(&elapsed_time_end , NULL);
    double elapsed_time = 0.001 * time_diff_in_ms( elapsed_time_start , elapsed_time_end );

    std::cout << "Finished " << n+1 << " steps for N=" << N << ", J=" << J << ", q=" << q/N;
    std::cout << " in " << elapsed_time << " seconds" << std::endl;
}
